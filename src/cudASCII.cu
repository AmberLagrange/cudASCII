#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void vectorAdd(int *a, int *b, int *c) {

        int i = threadIdx.x;
        c[i] = a[i] + b[i];

        return;
}

int main() {

        int a[] = { 1, 2, 3};
        int b[] = { 4, 5, 6};
        int c[sizeof(a) / sizeof(a[0])] = { 0 };

        int *cudaA = 0;
        int *cudaB = 0;
        int *cudaC = 0;

        hipMalloc(&cudaA, sizeof(a));
        hipMalloc(&cudaB, sizeof(b));
        hipMalloc(&cudaC, sizeof(c));

        hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
        hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

        vectorAdd<<< 1, sizeof(a) / sizeof(a[0]) >>>(cudaA, cudaB, cudaC);

        hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

        hipFree(cudaA);
        hipFree(cudaB);
        hipFree(cudaC);

        for (int i = 0; i < sizeof(c) / sizeof(c[0]); ++i) {
                printf("%d\n", c[i]);
        }

        return 0;
}