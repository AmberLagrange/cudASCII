#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "constants.h"

#include "image.h"
#include "ascii.h"

__global__ void ascii(char *greyscale, struct image_t *image, struct ascii_t *ascii) {

    int luminance = 0; // temp
    ascii->data[threadIdx.x * ascii->width + blockIdx.x] = greyscale[luminance];

    return;
}

__host__ int test_blank(struct ascii_t *ascii, const char *greyscale) {

    for (int i = 0; i < ascii->width * ascii->height; ++i) {
        if (ascii->data[i] != greyscale[0]) {
            printf("Error! Did not copy over properly!\nIndex %d has value %d\n", i, ascii->data[i]);
            return -1;
        }
    }

    printf("All values were copied over successfully!\n");

    return 0;
}

int main(int argc, char **argv) {

    if (argc < 2) {
        printf("Did not provide a filepath\n");
        return -1;
    }

    // Initial data on host

    // Host greyscale data
    const char *h_greyscale = GREYSCALE;

    // Host image data
    struct image_t h_image;
    read_image(&h_image, argv[1]);

    // Host ascii data
    struct ascii_t h_ascii;
    init_ascii(&h_ascii, h_image.width, SCALE_WIDTH, h_image.height, SCALE_HEIGHT);

    // Pointers to data on device
    char       *d_greyscale;
    struct image_t *d_image;
    struct ascii_t *d_ascii;

    // Create and copy greyscale data over
    hipHostMalloc(&d_greyscale, NUM_ASCII);
    hipMemcpy(d_greyscale, h_greyscale , NUM_ASCII, hipMemcpyHostToDevice);

    // Create and copy image struct over
    hipHostMalloc(&d_image, sizeof(struct image_t));
    hipMemcpy(d_image, &h_image, sizeof(h_image), hipMemcpyHostToDevice);

    // Create and copy image data over
    hipHostMalloc(&(d_image->data), d_image->width * d_image->height);
    hipMemcpy(d_image->data, h_image.data, d_image->width * d_image->height, hipMemcpyHostToDevice);

    // Create and copy ascii struct data
    hipHostMalloc(&d_ascii, sizeof(struct ascii_t));
    hipMemcpy(d_ascii, &h_ascii, sizeof(struct ascii_t), hipMemcpyHostToDevice);

    // Create ascii data
    hipHostMalloc(&(d_ascii->data), d_ascii->width * d_ascii->height);

    // Run the kernel
    ascii<<<h_ascii.width, h_ascii.height>>>(d_greyscale, d_image, d_ascii);

    // Copy ascii data from device to host
    hipMemcpy(h_ascii.data, d_ascii->data, h_ascii.width * h_ascii.height, hipMemcpyDeviceToHost);

    // Tests
    test_blank(&h_ascii, h_greyscale);

    // Clean up cuda memory
    hipHostFree(d_ascii->data);
    hipHostFree(d_ascii);
    hipHostFree(d_image->data);
    hipHostFree(d_image);
    hipHostFree(d_greyscale);

    // Clean up host memory
    cleanup_ascii(&h_ascii);
    cleanup_image(&h_image);
    
    hipDeviceReset();

    return 0;
}
